
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#define BLOCK_SIZE 16

#define for_x for (int x = 0; x < w; x++)
#define for_y for (int y = 0; y < h; y++)
#define for_xy for_x for_y

void init(unsigned *u, int w, int h) {
    for_xy u[y*w + x] = rand() < RAND_MAX / 10 ? 1 : 0;
}

__global__ void evolve_parallel(unsigned *univ, unsigned *new_arr, int w, int h) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int n = 0;
    for (int y1 = y - 1; y1 <= y + 1; y1++)
        for (int x1 = x - 1; x1 <= x + 1; x1++)
            if (univ[((y1 + h) % h) * w + ((x1 + w) % w)]) n++;

    if (univ[y*w + x]) n--;
    new_arr[y*w + x] = (n == 3 || (n == 2 && univ[y*w + x]));

}

void evolve(unsigned *univ, int w, int h) {
    
    unsigned* new_arr = (unsigned *)malloc(h * w * sizeof(unsigned));

    for (int y = 0; y < h; y++) {
        for (int x = 0; x < w; x++) {

            int n = 0;
            for (int y1 = y - 1; y1 <= y + 1; y1++)
                for (int x1 = x - 1; x1 <= x + 1; x1++)
                    if (univ[((y1 + h) % h) * w + ((x1 + w) % w)]) n++;

            if (univ[y*w + x]) n--;
            new_arr[y*w + x] = (n == 3 || (n == 2 && univ[y*w + x]));

        }
    }
    for (int y = 0; y < h; y++) {
        for (int x = 0; x < w; x++) {
            univ[y*w + x] = new_arr[y*w + x];
        }
    }
}

void game(unsigned *u, int w, int h, int iter) {
    hipEvent_t start_e = hipEvent_t();
    hipEvent_t stop_e = hipEvent_t();
    hipEventCreate(&start_e);
    hipEventCreate(&stop_e);

    size_t matrix_size = w * h * sizeof(unsigned);
    unsigned* current_matrix; 
    unsigned* next_matrix;

    hipMalloc(&current_matrix, matrix_size);
    hipMalloc(&next_matrix, matrix_size);

    hipMemcpy(current_matrix, u, matrix_size, hipMemcpyHostToDevice);

    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_size((w + BLOCK_SIZE - 1) / BLOCK_SIZE, (h + BLOCK_SIZE - 1) / BLOCK_SIZE);


    for (int i = 0; i < iter; i++) {
        if(i == iter / 2) {
           hipEventRecord(start_e, 0);
        }

        evolve_parallel<<<grid_size, block_size>>>(current_matrix, next_matrix, w, h);

        hipDeviceSynchronize();

        unsigned* tmp_matrix = current_matrix;
        current_matrix = next_matrix;
        next_matrix = tmp_matrix;


        if(i == iter / 2) {
            float elapsed_e = 0.f;
            hipEventRecord(stop_e, 0);
            hipEventSynchronize(stop_e);
            hipEventElapsedTime(&elapsed_e, start_e, stop_e);
            printf("Evolve time [ms] > %f \n\n", elapsed_e);

        }

    }

    hipMemcpy(u, current_matrix, matrix_size, hipMemcpyDeviceToHost);
    hipFree(current_matrix);
    hipFree(next_matrix);

	hipEventDestroy(start_e);
	hipEventDestroy(stop_e);    
}

int main(int c, char *v[]) {

    hipEvent_t start = hipEvent_t();
	hipEvent_t stop = hipEvent_t();
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);


    int w = 0, h = 0, iter = 0;
    unsigned *u;

    if (c > 1) w = atoi(v[1]);
    if (c > 2) h = atoi(v[2]);
    if (c > 3) iter = atoi(v[3]);
    if (w <= 0) w = 30;
    if (h <= 0) h = 30;
    if (iter <= 0) iter = 1000;

    u = (unsigned *)malloc(w * h * sizeof(unsigned));
    if (!u) exit(1);

    init(u, w, h);

    game(u, w, h, iter);

    free(u);

    float elapsed = 0.f;
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    printf("Total simulation time [ms] > %f \n\n", elapsed);

	hipEventDestroy(start);
	hipEventDestroy(stop);
}
